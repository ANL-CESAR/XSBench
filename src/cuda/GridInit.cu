#include "XSbench_header.cuh"

// Moves all required data structures to the GPU's memory space
SimulationData move_simulation_data_to_device( Inputs in, int mype, SimulationData SD )
{
	if(mype == 0) printf("Allocating and moving simulation data to GPU memory space...\n");

	////////////////////////////////////////////////////////////////////////////////
	// SUMMARY: Simulation Data Structure Manifest for "SD" Object
	// Here we list all heap arrays (and lengths) in SD that would need to be
	// offloaded manually if using an accelerator with a seperate memory space
	////////////////////////////////////////////////////////////////////////////////
	// int * num_nucs;                     // Length = length_num_nucs;
	// double * concs;                     // Length = length_concs
	// int * mats;                         // Length = length_mats
	// double * unionized_energy_array;    // Length = length_unionized_energy_array
	// int * index_grid;                   // Length = length_index_grid
	// NuclideGridPoint * nuclide_grid;    // Length = length_nuclide_grid
	// 
	// Note: "unionized_energy_array" and "index_grid" can be of zero length
	//        depending on lookup method.
	//
	// Note: "Lengths" are given as the number of objects in the array, not the
	//       number of bytes.
	////////////////////////////////////////////////////////////////////////////////
	size_t sz;
	size_t total_sz = 0;

	// Shallow copy of CPU simulation data to GPU simulation data
	SimulationData GSD = SD;

	// Move data to GPU memory space
	sz = GSD.length_num_nucs * sizeof(int);
	gpuErrchk( hipMalloc((void **) &GSD.num_nucs, sz) );
	gpuErrchk( hipMemcpy(GSD.num_nucs, SD.num_nucs, sz, hipMemcpyHostToDevice) );
	total_sz += sz;

	sz = GSD.length_concs * sizeof(double);
	gpuErrchk( hipMalloc((void **) &GSD.concs, sz) );
	gpuErrchk( hipMemcpy(GSD.concs, SD.concs, sz, hipMemcpyHostToDevice) );
	total_sz += sz;

	sz = GSD.length_mats * sizeof(int);
	gpuErrchk( hipMalloc((void **) &GSD.mats, sz) );
	gpuErrchk( hipMemcpy(GSD.mats, SD.mats, sz, hipMemcpyHostToDevice) );
	total_sz += sz;
	
	sz = GSD.length_unionized_energy_array * sizeof(double);
	gpuErrchk( hipMalloc((void **) &GSD.unionized_energy_array, sz) );
	gpuErrchk( hipMemcpy(GSD.unionized_energy_array, SD.unionized_energy_array, sz, hipMemcpyHostToDevice) );
	total_sz += sz;

	sz = GSD.length_index_grid * sizeof(int);
	gpuErrchk( hipMalloc((void **) &GSD.index_grid, sz) );
	gpuErrchk( hipMemcpy(GSD.index_grid, SD.index_grid, sz, hipMemcpyHostToDevice) );
	total_sz += sz;

	sz = GSD.length_nuclide_grid * sizeof(NuclideGridPoint);
	gpuErrchk( hipMalloc((void **) &GSD.nuclide_grid, sz) );
	gpuErrchk( hipMemcpy(GSD.nuclide_grid, SD.nuclide_grid, sz, hipMemcpyHostToDevice) );
	total_sz += sz;
	
	// Allocate verification array on device. This structure is not needed on CPU, so we don't
	// have to copy anything over.
	sz = in.lookups * sizeof(unsigned long);
	gpuErrchk( hipMalloc((void **) &GSD.verification, sz) );
	total_sz += sz;
	GSD.length_verification = in.lookups;
	
	// Synchronize
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	
	if(mype == 0 ) printf("GPU Intialization complete. Allocated %.0lf MB of data on GPU.\n", total_sz/1024.0/1024.0 );

	return GSD;

}

SimulationData grid_init_do_not_profile( Inputs in, int mype )
{
	// Structure to hold all allocated simuluation data arrays
	SimulationData SD;

	// Keep track of how much data we're allocating
	size_t nbytes = 0;
	
	// Set the initial seed value
	uint64_t seed = 42;	

	////////////////////////////////////////////////////////////////////
	// Initialize Nuclide Grids
	////////////////////////////////////////////////////////////////////
	
	if(mype == 0) printf("Intializing nuclide grids...\n");

	// First, we need to initialize our nuclide grid. This comes in the form
	// of a flattened 2D array that hold all the information we need to define
	// the cross sections for all isotopes in the simulation. 
	// The grid is composed of "NuclideGridPoint" structures, which hold the
	// energy level of the grid point and all associated XS data at that level.
	// An array of structures (AOS) is used instead of
	// a structure of arrays, as the grid points themselves are accessed in 
	// a random order, but all cross section interaction channels and the
	// energy level are read whenever the gridpoint is accessed, meaning the
	// AOS is more cache efficient.
	
	// Initialize Nuclide Grid
	SD.length_nuclide_grid = in.n_isotopes * in.n_gridpoints;
	SD.nuclide_grid     = (NuclideGridPoint *) malloc( SD.length_nuclide_grid * sizeof(NuclideGridPoint));
	assert(SD.nuclide_grid != NULL);
	nbytes += SD.length_nuclide_grid * sizeof(NuclideGridPoint);
	for( int i = 0; i < SD.length_nuclide_grid; i++ )
	{
		SD.nuclide_grid[i].energy        = LCG_random_double(&seed);
		SD.nuclide_grid[i].total_xs      = LCG_random_double(&seed);
		SD.nuclide_grid[i].elastic_xs    = LCG_random_double(&seed);
		SD.nuclide_grid[i].absorbtion_xs = LCG_random_double(&seed);
		SD.nuclide_grid[i].fission_xs    = LCG_random_double(&seed);
		SD.nuclide_grid[i].nu_fission_xs = LCG_random_double(&seed);
	}

	// Sort so that each nuclide has data stored in ascending energy order.
	for( int i = 0; i < in.n_isotopes; i++ )
		qsort( &SD.nuclide_grid[i*in.n_gridpoints], in.n_gridpoints, sizeof(NuclideGridPoint), NGP_compare);
	
	// error debug check
	/*
	for( int i = 0; i < in.n_isotopes; i++ )
	{
		printf("NUCLIDE %d ==============================\n", i);
		for( int j = 0; j < in.n_gridpoints; j++ )
			printf("E%d = %lf\n", j, SD.nuclide_grid[i * in.n_gridpoints + j].energy);
	}
	*/
	

	////////////////////////////////////////////////////////////////////
	// Initialize Acceleration Structure
	////////////////////////////////////////////////////////////////////
	
	if( in.grid_type == NUCLIDE )
	{
		SD.length_unionized_energy_array = 0;
		SD.length_index_grid = 0;
	}
	
	if( in.grid_type == UNIONIZED )
	{
		if(mype == 0) printf("Intializing unionized grid...\n");

		// Allocate space to hold the union of all nuclide energy data
		SD.length_unionized_energy_array = in.n_isotopes * in.n_gridpoints;
		SD.unionized_energy_array = (double *) malloc( SD.length_unionized_energy_array * sizeof(double));
		assert(SD.unionized_energy_array != NULL );
		nbytes += SD.length_unionized_energy_array * sizeof(double);

		// Copy energy data over from the nuclide energy grid
		for( int i = 0; i < SD.length_unionized_energy_array; i++ )
			SD.unionized_energy_array[i] = SD.nuclide_grid[i].energy;

		// Sort unionized energy array
		qsort( SD.unionized_energy_array, SD.length_unionized_energy_array, sizeof(double), double_compare);

		// Allocate space to hold the acceleration grid indices
		SD.length_index_grid = SD.length_unionized_energy_array * in.n_isotopes;
		SD.index_grid = (int *) malloc( SD.length_index_grid * sizeof(int));
		assert(SD.index_grid != NULL);
		nbytes += SD.length_index_grid * sizeof(int);

		// Generates the double indexing grid
		int * idx_low = (int *) calloc( in.n_isotopes, sizeof(int));
		assert(idx_low != NULL );
		double * energy_high = (double *) malloc( in.n_isotopes * sizeof(double));
		assert(energy_high != NULL );

		for( int i = 0; i < in.n_isotopes; i++ )
			energy_high[i] = SD.nuclide_grid[i * in.n_gridpoints + 1].energy;

		for( long e = 0; e < SD.length_unionized_energy_array; e++ )
		{
			double unionized_energy = SD.unionized_energy_array[e];
			for( long i = 0; i < in.n_isotopes; i++ )
			{
				if( unionized_energy < energy_high[i]  )
					SD.index_grid[e * in.n_isotopes + i] = idx_low[i];
				else if( idx_low[i] == in.n_gridpoints - 2 )
					SD.index_grid[e * in.n_isotopes + i] = idx_low[i];
				else
				{
					idx_low[i]++;
					SD.index_grid[e * in.n_isotopes + i] = idx_low[i];
					energy_high[i] = SD.nuclide_grid[i * in.n_gridpoints + idx_low[i] + 1].energy;	
				}
			}
		}

		free(idx_low);
		free(energy_high);
	}

	if( in.grid_type == HASH )
	{
		if(mype == 0) printf("Intializing hash grid...\n");
		SD.length_unionized_energy_array = 0;
		SD.length_index_grid  = in.hash_bins * in.n_isotopes;
		SD.index_grid = (int *) malloc( SD.length_index_grid * sizeof(int)); 
		assert(SD.index_grid != NULL);
		nbytes += SD.length_index_grid * sizeof(int);

		double du = 1.0 / in.hash_bins;

		// For each energy level in the hash table
		for( long e = 0; e < in.hash_bins; e++ )
		{
			double energy = e * du;

			// We need to determine the bounding energy levels for all isotopes
			for( long i = 0; i < in.n_isotopes; i++ )
			{
				SD.index_grid[e * in.n_isotopes + i] = grid_search_nuclide( in.n_gridpoints, energy, SD.nuclide_grid + i * in.n_gridpoints, 0, in.n_gridpoints-1);
			}
		}
	}

	////////////////////////////////////////////////////////////////////
	// Initialize Materials and Concentrations
	////////////////////////////////////////////////////////////////////
	if(mype == 0) printf("Intializing material data...\n");
	
	// Set the number of nuclides in each material
	SD.num_nucs  = load_num_nucs(in.n_isotopes);
	SD.length_num_nucs = 12; // There are always 12 materials in XSBench

	// Intialize the flattened 2D grid of material data. The grid holds
	// a list of nuclide indices for each of the 12 material types. The
	// grid is allocated as a full square grid, even though not all
	// materials have the same number of nuclides.
	SD.mats = load_mats(SD.num_nucs, in.n_isotopes, &SD.max_num_nucs);
	SD.length_mats = SD.length_num_nucs * SD.max_num_nucs;

	// Intialize the flattened 2D grid of nuclide concentration data. The grid holds
	// a list of nuclide concentrations for each of the 12 material types. The
	// grid is allocated as a full square grid, even though not all
	// materials have the same number of nuclides.
	SD.concs = load_concs(SD.num_nucs, SD.max_num_nucs);
	SD.length_concs = SD.length_mats;

	if(mype == 0) printf("Intialization complete. Allocated %.0lf MB of data on CPU.\n", nbytes/1024.0/1024.0 );

	return SD;
}
