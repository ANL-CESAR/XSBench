#include "hip/hip_runtime.h"
#include "XSbench_header.cuh"

////////////////////////////////////////////////////////////////////////////////////
// BASELINE FUNCTIONS
////////////////////////////////////////////////////////////////////////////////////
// All "baseline" code is at the top of this file. The baseline code is a simple
// port of the original CPU OpenMP code to CUDA with few significant changes or
// optimizations made. Following these functions are a number of optimized variants,
// which each deploy a different combination of optimizations strategies. By
// default, XSBench will only run the baseline implementation. Optimized variants
// must be specifically selected using the "-k <optimized variant ID>" command
// line argument.
////////////////////////////////////////////////////////////////////////////////////

unsigned long long run_event_based_simulation_baseline(Inputs in, SimulationData GSD, int mype)
{
	////////////////////////////////////////////////////////////////////////////////
	// Configure & Launch Simulation Kernel
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Running baseline event-based simulation...\n");

	int nthreads = 32;
	int nblocks = ceil( (double) in.lookups / 32.0);

	xs_lookup_kernel_baseline<<<nblocks, nthreads>>>( in, GSD );
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	
	////////////////////////////////////////////////////////////////////////////////
	// Reduce Verification Results
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Reducing verification results...\n");

	unsigned long verification_scalar = thrust::reduce(thrust::device, GSD.verification, GSD.verification + in.lookups, 0);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	return verification_scalar;
}

// In this kernel, we perform a single lookup with each thread. Threads within a warp
// do not really have any relation to each other, and divergence due to high nuclide count fuel
// material lookups are costly. This kernel constitutes baseline performance.
__global__ void xs_lookup_kernel_baseline(Inputs in, SimulationData GSD )
{
	// The lookup ID. Used to set the seed, and to store the verification value
	const int i = blockIdx.x *blockDim.x + threadIdx.x;

	if( i >= in.lookups )
		return;

	// Set the initial seed value
	uint64_t seed = STARTING_SEED;	

	// Forward seed to lookup index (we need 2 samples per lookup)
	seed = fast_forward_LCG(seed, 2*i);

	// Randomly pick an energy and material for the particle
	double p_energy = LCG_random_double(&seed);
	int mat         = pick_mat(&seed); 
		
	double macro_xs_vector[5] = {0};
		
	// Perform macroscopic Cross Section Lookup
	calculate_macro_xs(
			p_energy,        // Sampled neutron energy (in lethargy)
			mat,             // Sampled material type index neutron is in
			in.n_isotopes,   // Total number of isotopes in simulation
			in.n_gridpoints, // Number of gridpoints per isotope in simulation
			GSD.num_nucs,     // 1-D array with number of nuclides per material
			GSD.concs,        // Flattened 2-D array with concentration of each nuclide in each material
			GSD.unionized_energy_array, // 1-D Unionized energy array
			GSD.index_grid,   // Flattened 2-D grid holding indices into nuclide grid for each unionized energy level
			GSD.nuclide_grid, // Flattened 2-D grid holding energy levels and XS_data for all nuclides in simulation
			GSD.mats,         // Flattened 2-D array with nuclide indices defining composition of each type of material
			macro_xs_vector, // 1-D array with result of the macroscopic cross section (5 different reaction channels)
			in.grid_type,    // Lookup type (nuclide, hash, or unionized)
			in.hash_bins,    // Number of hash bins used (if using hash lookup type)
			GSD.max_num_nucs  // Maximum number of nuclides present in any material
			);

	// For verification, and to prevent the compiler from optimizing
	// all work out, we interrogate the returned macro_xs_vector array
	// to find its maximum value index, then increment the verification
	// value by that index. In this implementation, we have each thread
	// write to its thread_id index in an array, which we will reduce
	// with a thrust reduction kernel after the main simulation kernel.
	double max = -1.0;
	int max_idx = 0;
	for(int j = 0; j < 5; j++ )
	{
		if( macro_xs_vector[j] > max )
		{
			max = macro_xs_vector[j];
			max_idx = j;
		}
	}
	GSD.verification[i] = max_idx+1;
}

// Calculates the microscopic cross section for a given nuclide & energy
__device__ void calculate_micro_xs(   double p_energy, int nuc, long n_isotopes,
                           long n_gridpoints,
                           double * __restrict__ egrid, int * __restrict__ index_data,
                           NuclideGridPoint * __restrict__ nuclide_grids,
                           long idx, double * __restrict__ xs_vector, int grid_type, int hash_bins ){
	// Variables
	double f;
	NuclideGridPoint * low, * high;

	// If using only the nuclide grid, we must perform a binary search
	// to find the energy location in this particular nuclide's grid.
	if( grid_type == NUCLIDE )
	{
		// Perform binary search on the Nuclide Grid to find the index
		idx = grid_search_nuclide( n_gridpoints, p_energy, &nuclide_grids[nuc*n_gridpoints], 0, n_gridpoints-1);

		// pull ptr from nuclide grid and check to ensure that
		// we're not reading off the end of the nuclide's grid
		if( idx == n_gridpoints - 1 )
			low = &nuclide_grids[nuc*n_gridpoints + idx - 1];
		else
			low = &nuclide_grids[nuc*n_gridpoints + idx];
	}
	else if( grid_type == UNIONIZED) // Unionized Energy Grid - we already know the index, no binary search needed.
	{
		// pull ptr from energy grid and check to ensure that
		// we're not reading off the end of the nuclide's grid
		if( index_data[idx * n_isotopes + nuc] == n_gridpoints - 1 )
			low = &nuclide_grids[nuc*n_gridpoints + index_data[idx * n_isotopes + nuc] - 1];
		else
			low = &nuclide_grids[nuc*n_gridpoints + index_data[idx * n_isotopes + nuc]];
	}
	else // Hash grid
	{
		// load lower bounding index
		int u_low = index_data[idx * n_isotopes + nuc];

		// Determine higher bounding index
		int u_high;
		if( idx == hash_bins - 1 )
			u_high = n_gridpoints - 1;
		else
			u_high = index_data[(idx+1)*n_isotopes + nuc] + 1;

		// Check edge cases to make sure energy is actually between these
		// Then, if things look good, search for gridpoint in the nuclide grid
		// within the lower and higher limits we've calculated.
		double e_low  = nuclide_grids[nuc*n_gridpoints + u_low].energy;
		double e_high = nuclide_grids[nuc*n_gridpoints + u_high].energy;
		int lower;
		if( p_energy <= e_low )
			lower = 0;
		else if( p_energy >= e_high )
			lower = n_gridpoints - 1;
		else
			lower = grid_search_nuclide( n_gridpoints, p_energy, &nuclide_grids[nuc*n_gridpoints], u_low, u_high);

		if( lower == n_gridpoints - 1 )
			low = &nuclide_grids[nuc*n_gridpoints + lower - 1];
		else
			low = &nuclide_grids[nuc*n_gridpoints + lower];
	}
	
	high = low + 1;
	
	// calculate the re-useable interpolation factor
	f = (high->energy - p_energy) / (high->energy - low->energy);

	// Total XS
	xs_vector[0] = high->total_xs - f * (high->total_xs - low->total_xs);
	
	// Elastic XS
	xs_vector[1] = high->elastic_xs - f * (high->elastic_xs - low->elastic_xs);
	
	// Absorbtion XS
	xs_vector[2] = high->absorbtion_xs - f * (high->absorbtion_xs - low->absorbtion_xs);
	
	// Fission XS
	xs_vector[3] = high->fission_xs - f * (high->fission_xs - low->fission_xs);
	
	// Nu Fission XS
	xs_vector[4] = high->nu_fission_xs - f * (high->nu_fission_xs - low->nu_fission_xs);
}

// Calculates macroscopic cross section based on a given material & energy 
__device__ void calculate_macro_xs( double p_energy, int mat, long n_isotopes,
                         long n_gridpoints, int * __restrict__ num_nucs,
                         double * __restrict__ concs,
                         double * __restrict__ egrid, int * __restrict__ index_data,
                         NuclideGridPoint * __restrict__ nuclide_grids,
                         int * __restrict__ mats,
                         double * __restrict__ macro_xs_vector, int grid_type, int hash_bins, int max_num_nucs ){
	int p_nuc; // the nuclide we are looking up
	long idx = -1;	
	double conc; // the concentration of the nuclide in the material

	// cleans out macro_xs_vector
	for( int k = 0; k < 5; k++ )
		macro_xs_vector[k] = 0;

	// If we are using the unionized energy grid (UEG), we only
	// need to perform 1 binary search per macroscopic lookup.
	// If we are using the nuclide grid search, it will have to be
	// done inside of the "calculate_micro_xs" function for each different
	// nuclide in the material.
	if( grid_type == UNIONIZED )
		idx = grid_search( n_isotopes * n_gridpoints, p_energy, egrid);	
	else if( grid_type == HASH )
	{
		double du = 1.0 / hash_bins;
		idx = p_energy / du;
	}
	
	// Once we find the pointer array on the UEG, we can pull the data
	// from the respective nuclide grids, as well as the nuclide
	// concentration data for the material
	// Each nuclide from the material needs to have its micro-XS array
	// looked up & interpolatied (via calculate_micro_xs). Then, the
	// micro XS is multiplied by the concentration of that nuclide
	// in the material, and added to the total macro XS array.
	// (Independent -- though if parallelizing, must use atomic operations
	//  or otherwise control access to the xs_vector and macro_xs_vector to
	//  avoid simulataneous writing to the same data structure)
	for( int j = 0; j < num_nucs[mat]; j++ )
	{
		double xs_vector[5];
		p_nuc = mats[mat*max_num_nucs + j];
		conc = concs[mat*max_num_nucs + j];
		calculate_micro_xs( p_energy, p_nuc, n_isotopes,
		                    n_gridpoints, egrid, index_data,
		                    nuclide_grids, idx, xs_vector, grid_type, hash_bins );
		for( int k = 0; k < 5; k++ )
			macro_xs_vector[k] += xs_vector[k] * conc;
	}
}


// binary search for energy on unionized energy grid
// returns lower index
__device__ long grid_search( long n, double quarry, double * __restrict__ A)
{
	long lowerLimit = 0;
	long upperLimit = n-1;
	long examinationPoint;
	long length = upperLimit - lowerLimit;

	while( length > 1 )
	{
		examinationPoint = lowerLimit + ( length / 2 );
		
		if( A[examinationPoint] > quarry )
			upperLimit = examinationPoint;
		else
			lowerLimit = examinationPoint;
		
		length = upperLimit - lowerLimit;
	}
	
	return lowerLimit;
}

// binary search for energy on nuclide energy grid
__host__ __device__ long grid_search_nuclide( long n, double quarry, NuclideGridPoint * A, long low, long high)
{
	long lowerLimit = low;
	long upperLimit = high;
	long examinationPoint;
	long length = upperLimit - lowerLimit;

	while( length > 1 )
	{
		examinationPoint = lowerLimit + ( length / 2 );
		
		if( A[examinationPoint].energy > quarry )
			upperLimit = examinationPoint;
		else
			lowerLimit = examinationPoint;
		
		length = upperLimit - lowerLimit;
	}
	
	return lowerLimit;
}

// picks a material based on a probabilistic distribution
__device__ int pick_mat( uint64_t * seed )
{
	// I have a nice spreadsheet supporting these numbers. They are
	// the fractions (by volume) of material in the core. Not a 
	// *perfect* approximation of where XS lookups are going to occur,
	// but this will do a good job of biasing the system nonetheless.

	// Also could be argued that doing fractions by weight would be 
	// a better approximation, but volume does a good enough job for now.

	double dist[12];
	dist[0]  = 0.140;	// fuel
	dist[1]  = 0.052;	// cladding
	dist[2]  = 0.275;	// cold, borated water
	dist[3]  = 0.134;	// hot, borated water
	dist[4]  = 0.154;	// RPV
	dist[5]  = 0.064;	// Lower, radial reflector
	dist[6]  = 0.066;	// Upper reflector / top plate
	dist[7]  = 0.055;	// bottom plate
	dist[8]  = 0.008;	// bottom nozzle
	dist[9]  = 0.015;	// top nozzle
	dist[10] = 0.025;	// top of fuel assemblies
	dist[11] = 0.013;	// bottom of fuel assemblies
	
	double roll = LCG_random_double(seed);

	// makes a pick based on the distro
	for( int i = 0; i < 12; i++ )
	{
		double running = 0;
		for( int j = i; j > 0; j-- )
			running += dist[j];
		if( roll < running )
			return i;
	}

	return 0;
}

__host__ __device__ double LCG_random_double(uint64_t * seed)
{
	// LCG parameters
	const uint64_t m = 9223372036854775808ULL; // 2^63
	const uint64_t a = 2806196910506780709ULL;
	const uint64_t c = 1ULL;
	*seed = (a * (*seed) + c) % m;
	return (double) (*seed) / (double) m;
}	

__device__ uint64_t fast_forward_LCG(uint64_t seed, uint64_t n)
{
	// LCG parameters
	const uint64_t m = 9223372036854775808ULL; // 2^63
	uint64_t a = 2806196910506780709ULL;
	uint64_t c = 1ULL;

	n = n % m;

	uint64_t a_new = 1;
	uint64_t c_new = 0;

	while(n > 0) 
	{
		if(n & 1)
		{
			a_new *= a;
			c_new = c_new * a + c;
		}
		c *= (a + 1);
		a *= a;

		n >>= 1;
	}

	return (a_new * seed + c_new) % m;
}

////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////
// OPTIMIZED VARIANT FUNCTIONS
////////////////////////////////////////////////////////////////////////////////////
// This section contains a number of optimized variants of some of the above
// functions, which each deploy a different combination of optimizations strategies
// specific to GPU. By default, XSBench will not run any of these variants. They
// must be specifically selected using the "-k <optimized variant ID>" command
// line argument.
////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////////
// Optimization 1 -- Basic kernel splitting of sampling & lookup routines
////////////////////////////////////////////////////////////////////////////////////
// This optimization requires a little extra data to store all material IDs and 
// energies for the sampled particles between kernel calls. By itself, this
// optimization is likely actually a bit of a slowdown compared to the baseline
// kernel. However, it will be used by better optimization kernels down the line.
////////////////////////////////////////////////////////////////////////////////////
unsigned long long run_event_based_simulation_optimization_1(Inputs in, SimulationData GSD, int mype)
{
	const char * optimization_name = "Optimization 1 - basic sample/lookup kernel splitting";
	
	if( mype == 0)	printf("Simulation Kernel:\"%s\"\n", optimization_name);
	
	////////////////////////////////////////////////////////////////////////////////
	// Allocate Additional Data Structures Needed by Optimized Kernel
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Allocating additional device data required by kernel...\n");
	size_t sz;
	size_t total_sz = 0;

	sz = in.lookups * sizeof(double);
	gpuErrchk( hipMalloc((void **) &GSD.p_energy_samples, sz) );
	total_sz += sz;
	GSD.length_p_energy_samples = in.lookups;

	sz = in.lookups * sizeof(int);
	gpuErrchk( hipMalloc((void **) &GSD.mat_samples, sz) );
	total_sz += sz;
	GSD.length_mat_samples = in.lookups;
	
	if( mype == 0)	printf("Allocated an additional %.0lf MB of data on GPU.\n", total_sz/1024.0/1024.0);

	////////////////////////////////////////////////////////////////////////////////
	// Configure & Launch Simulation Kernel
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Beginning optimized simulation...\n");

	int nthreads = 32;
	int nblocks = ceil( (double) in.lookups / 32.0);
	
	sampling_kernel<<<nblocks, nthreads>>>( in, GSD );
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	xs_lookup_kernel_optimization_1<<<nblocks, nthreads>>>( in, GSD );
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	
	////////////////////////////////////////////////////////////////////////////////
	// Reduce Verification Results
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Reducing verification results...\n");

	unsigned long verification_scalar = thrust::reduce(thrust::device, GSD.verification, GSD.verification + in.lookups, 0);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	return verification_scalar;
}

__global__ void sampling_kernel(Inputs in, SimulationData GSD )
{
	// The lookup ID.
	const int i = blockIdx.x *blockDim.x + threadIdx.x;

	if( i >= in.lookups )
		return;

	// Set the initial seed value
	uint64_t seed = STARTING_SEED;	

	// Forward seed to lookup index (we need 2 samples per lookup)
	seed = fast_forward_LCG(seed, 2*i);

	// Randomly pick an energy and material for the particle
	double p_energy = LCG_random_double(&seed);
	int mat         = pick_mat(&seed); 

	// Store sample data in state array
	GSD.p_energy_samples[i] = p_energy;
	GSD.mat_samples[i] = mat;
}

__global__ void xs_lookup_kernel_optimization_1(Inputs in, SimulationData GSD )
{
	// The lookup ID. Used to set the seed, and to store the verification value
	const int i = blockIdx.x *blockDim.x + threadIdx.x;

	if( i >= in.lookups )
		return;
		
	double macro_xs_vector[5] = {0};
		
	// Perform macroscopic Cross Section Lookup
	calculate_macro_xs(
			GSD.p_energy_samples[i],        // Sampled neutron energy (in lethargy)
			GSD.mat_samples[i],             // Sampled material type index neutron is in
			in.n_isotopes,   // Total number of isotopes in simulation
			in.n_gridpoints, // Number of gridpoints per isotope in simulation
			GSD.num_nucs,     // 1-D array with number of nuclides per material
			GSD.concs,        // Flattened 2-D array with concentration of each nuclide in each material
			GSD.unionized_energy_array, // 1-D Unionized energy array
			GSD.index_grid,   // Flattened 2-D grid holding indices into nuclide grid for each unionized energy level
			GSD.nuclide_grid, // Flattened 2-D grid holding energy levels and XS_data for all nuclides in simulation
			GSD.mats,         // Flattened 2-D array with nuclide indices defining composition of each type of material
			macro_xs_vector, // 1-D array with result of the macroscopic cross section (5 different reaction channels)
			in.grid_type,    // Lookup type (nuclide, hash, or unionized)
			in.hash_bins,    // Number of hash bins used (if using hash lookup type)
			GSD.max_num_nucs  // Maximum number of nuclides present in any material
			);

	// For verification, and to prevent the compiler from optimizing
	// all work out, we interrogate the returned macro_xs_vector array
	// to find its maximum value index, then increment the verification
	// value by that index. In this implementation, we have each thread
	// write to its thread_id index in an array, which we will reduce
	// with a thrust reduction kernel after the main simulation kernel.
	double max = -1.0;
	int max_idx = 0;
	for(int j = 0; j < 5; j++ )
	{
		if( macro_xs_vector[j] > max )
		{
			max = macro_xs_vector[j];
			max_idx = j;
		}
	}
	GSD.verification[i] = max_idx+1;
}

////////////////////////////////////////////////////////////////////////////////////
// Optimization 2 -- Kernel Splitting + Material-Specific Lookup Kernels
////////////////////////////////////////////////////////////////////////////////////
// This one builds on the first optimization. It uses multiple kernels, one
// for each material type, to better balance the workload across threads within
// a warp. This works because each material will have a different number of 
// isotopes, with some having a ton, meaning that SIMD efficiency can be rather
// low by default. Better efficiency may be gained in further optimizations by
// sorting the lookups first.
////////////////////////////////////////////////////////////////////////////////////
unsigned long long run_event_based_simulation_optimization_2(Inputs in, SimulationData GSD, int mype)
{
	const char * optimization_name = "Optimization 2 - Material Lookup Kernels";
	
	if( mype == 0)	printf("Simulation Kernel:\"%s\"\n", optimization_name);
	
	////////////////////////////////////////////////////////////////////////////////
	// Allocate Additional Data Structures Needed by Optimized Kernel
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Allocating additional device data required by kernel...\n");
	size_t sz;
	size_t total_sz = 0;

	sz = in.lookups * sizeof(double);
	gpuErrchk( hipMalloc((void **) &GSD.p_energy_samples, sz) );
	total_sz += sz;
	GSD.length_p_energy_samples = in.lookups;

	sz = in.lookups * sizeof(int);
	gpuErrchk( hipMalloc((void **) &GSD.mat_samples, sz) );
	total_sz += sz;
	GSD.length_mat_samples = in.lookups;
	
	if( mype == 0)	printf("Allocated an additional %.0lf MB of data on GPU.\n", total_sz/1024.0/1024.0);

	////////////////////////////////////////////////////////////////////////////////
	// Configure & Launch Simulation Kernel
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Beginning optimized simulation...\n");

	int nthreads = 32;
	int nblocks = ceil( (double) in.lookups / 32.0);
	
	sampling_kernel<<<nblocks, nthreads>>>( in, GSD );
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	// Launch all material kernels individually
	for( int m = 0; m < 12; m++ )
		xs_lookup_kernel_optimization_2<<<nblocks, nthreads>>>( in, GSD, m );
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	
	////////////////////////////////////////////////////////////////////////////////
	// Reduce Verification Results
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Reducing verification results...\n");

	unsigned long verification_scalar = thrust::reduce(thrust::device, GSD.verification, GSD.verification + in.lookups, 0);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	return verification_scalar;
}

__global__ void xs_lookup_kernel_optimization_2(Inputs in, SimulationData GSD, int m )
{
	// The lookup ID. Used to set the seed, and to store the verification value
	const int i = blockIdx.x *blockDim.x + threadIdx.x;

	if( i >= in.lookups )
		return;
	
	// Check that our material type matches the kernel material
	int mat = GSD.mat_samples[i];
	if( mat != m )
		return;

	double macro_xs_vector[5] = {0};
		
	// Perform macroscopic Cross Section Lookup
	calculate_macro_xs(
			GSD.p_energy_samples[i],        // Sampled neutron energy (in lethargy)
			mat,             // Sampled material type index neutron is in
			in.n_isotopes,   // Total number of isotopes in simulation
			in.n_gridpoints, // Number of gridpoints per isotope in simulation
			GSD.num_nucs,     // 1-D array with number of nuclides per material
			GSD.concs,        // Flattened 2-D array with concentration of each nuclide in each material
			GSD.unionized_energy_array, // 1-D Unionized energy array
			GSD.index_grid,   // Flattened 2-D grid holding indices into nuclide grid for each unionized energy level
			GSD.nuclide_grid, // Flattened 2-D grid holding energy levels and XS_data for all nuclides in simulation
			GSD.mats,         // Flattened 2-D array with nuclide indices defining composition of each type of material
			macro_xs_vector, // 1-D array with result of the macroscopic cross section (5 different reaction channels)
			in.grid_type,    // Lookup type (nuclide, hash, or unionized)
			in.hash_bins,    // Number of hash bins used (if using hash lookup type)
			GSD.max_num_nucs  // Maximum number of nuclides present in any material
			);

	// For verification, and to prevent the compiler from optimizing
	// all work out, we interrogate the returned macro_xs_vector array
	// to find its maximum value index, then increment the verification
	// value by that index. In this implementation, we have each thread
	// write to its thread_id index in an array, which we will reduce
	// with a thrust reduction kernel after the main simulation kernel.
	double max = -1.0;
	int max_idx = 0;
	for(int j = 0; j < 5; j++ )
	{
		if( macro_xs_vector[j] > max )
		{
			max = macro_xs_vector[j];
			max_idx = j;
		}
	}
	GSD.verification[i] = max_idx+1;
}


////////////////////////////////////////////////////////////////////////////////////
// Optimization 3 -- Kernel Splitting + Fuel or Not-Fuel Lookups
////////////////////////////////////////////////////////////////////////////////////
// This optimization alters Optimization 2. Instead of executing a kernel call for
// ALL different material types, only two different calls are made. One for fuel,
// and one for all the other materials. As the fuel material has by far the most
// isotopes, it takes much longer than the rest.
////////////////////////////////////////////////////////////////////////////////////
unsigned long long run_event_based_simulation_optimization_3(Inputs in, SimulationData GSD, int mype)
{
	const char * optimization_name = "Optimization 3 - Fuel or Other Lookup Kernels";
	
	if( mype == 0)	printf("Simulation Kernel:\"%s\"\n", optimization_name);
	
	////////////////////////////////////////////////////////////////////////////////
	// Allocate Additional Data Structures Needed by Optimized Kernel
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Allocating additional device data required by kernel...\n");
	size_t sz;
	size_t total_sz = 0;

	sz = in.lookups * sizeof(double);
	gpuErrchk( hipMalloc((void **) &GSD.p_energy_samples, sz) );
	total_sz += sz;
	GSD.length_p_energy_samples = in.lookups;

	sz = in.lookups * sizeof(int);
	gpuErrchk( hipMalloc((void **) &GSD.mat_samples, sz) );
	total_sz += sz;
	GSD.length_mat_samples = in.lookups;
	
	if( mype == 0)	printf("Allocated an additional %.0lf MB of data on GPU.\n", total_sz/1024.0/1024.0);

	////////////////////////////////////////////////////////////////////////////////
	// Configure & Launch Simulation Kernel
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Beginning optimized simulation...\n");

	int nthreads = 32;
	int nblocks = ceil( (double) in.lookups / 32.0);
	
	sampling_kernel<<<nblocks, nthreads>>>( in, GSD );
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	// Launch all material kernels individually
	xs_lookup_kernel_optimization_3<<<nblocks, nthreads>>>( in, GSD, 0 );
	xs_lookup_kernel_optimization_3<<<nblocks, nthreads>>>( in, GSD, 1 );
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	
	////////////////////////////////////////////////////////////////////////////////
	// Reduce Verification Results
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Reducing verification results...\n");

	unsigned long verification_scalar = thrust::reduce(thrust::device, GSD.verification, GSD.verification + in.lookups, 0);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	return verification_scalar;
}

__global__ void xs_lookup_kernel_optimization_3(Inputs in, SimulationData GSD, int is_fuel )
{
	// The lookup ID. Used to set the seed, and to store the verification value
	const int i = blockIdx.x *blockDim.x + threadIdx.x;

	if( i >= in.lookups )
		return;
	
	int mat = GSD.mat_samples[i];

	// If this is the fuel kernel, AND this is a fuel lookup, then perform a lookup
	// OR if this is not the fuel kernel, AND this is not a fuel lookup, then perform the lookup
	if( ((is_fuel == 1) && (mat == 0)) || ((is_fuel == 0) && (mat != 0 ) ))
	{
		double macro_xs_vector[5] = {0};
			
		// Perform macroscopic Cross Section Lookup
		calculate_macro_xs(
				GSD.p_energy_samples[i],        // Sampled neutron energy (in lethargy)
				mat,             // Sampled material type index neutron is in
				in.n_isotopes,   // Total number of isotopes in simulation
				in.n_gridpoints, // Number of gridpoints per isotope in simulation
				GSD.num_nucs,     // 1-D array with number of nuclides per material
				GSD.concs,        // Flattened 2-D array with concentration of each nuclide in each material
				GSD.unionized_energy_array, // 1-D Unionized energy array
				GSD.index_grid,   // Flattened 2-D grid holding indices into nuclide grid for each unionized energy level
				GSD.nuclide_grid, // Flattened 2-D grid holding energy levels and XS_data for all nuclides in simulation
				GSD.mats,         // Flattened 2-D array with nuclide indices defining composition of each type of material
				macro_xs_vector, // 1-D array with result of the macroscopic cross section (5 different reaction channels)
				in.grid_type,    // Lookup type (nuclide, hash, or unionized)
				in.hash_bins,    // Number of hash bins used (if using hash lookup type)
				GSD.max_num_nucs  // Maximum number of nuclides present in any material
				);

		// For verification, and to prevent the compiler from optimizing
		// all work out, we interrogate the returned macro_xs_vector array
		// to find its maximum value index, then increment the verification
		// value by that index. In this implementation, we have each thread
		// write to its thread_id index in an array, which we will reduce
		// with a thrust reduction kernel after the main simulation kernel.
		double max = -1.0;
		int max_idx = 0;
		for(int j = 0; j < 5; j++ )
		{
			if( macro_xs_vector[j] > max )
			{
				max = macro_xs_vector[j];
				max_idx = j;
			}
		}
		GSD.verification[i] = max_idx+1;
	}
}


////////////////////////////////////////////////////////////////////////////////////
// Optimization 4 -- Kernel Splitting + All Material Lookups + Full Sort
////////////////////////////////////////////////////////////////////////////////////
// This optimization builds on optimization 2, adding in a full sort before
// hand so that the warps should be densely packed together. This should maximize
// SIMD efficiency of the kernel, but may incur an added cost for the sort.
////////////////////////////////////////////////////////////////////////////////////
unsigned long long run_event_based_simulation_optimization_4(Inputs in, SimulationData GSD, int mype)
{
	const char * optimization_name = "Optimization 4 - All Material Lookup Kernels + Material Sort";
	
	if( mype == 0)	printf("Simulation Kernel:\"%s\"\n", optimization_name);
	
	////////////////////////////////////////////////////////////////////////////////
	// Allocate Additional Data Structures Needed by Optimized Kernel
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Allocating additional device data required by kernel...\n");
	size_t sz;
	size_t total_sz = 0;

	sz = in.lookups * sizeof(double);
	gpuErrchk( hipMalloc((void **) &GSD.p_energy_samples, sz) );
	total_sz += sz;
	GSD.length_p_energy_samples = in.lookups;

	sz = in.lookups * sizeof(int);
	gpuErrchk( hipMalloc((void **) &GSD.mat_samples, sz) );
	total_sz += sz;
	GSD.length_mat_samples = in.lookups;
	
	if( mype == 0)	printf("Allocated an additional %.0lf MB of data on GPU.\n", total_sz/1024.0/1024.0);

	////////////////////////////////////////////////////////////////////////////////
	// Configure & Launch Simulation Kernel
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Beginning optimized simulation...\n");

	int nthreads = 32;
	int nblocks = ceil( (double) in.lookups / 32.0);
	
	sampling_kernel<<<nblocks, nthreads>>>( in, GSD );
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	// Count the number of fuel material lookups that need to be performed (fuel id = 0)
	int n_lookups_per_material[12];
	for( int m = 0; m < 12; m++ )
		n_lookups_per_material[m] = thrust::count(thrust::device, GSD.mat_samples, GSD.mat_samples + in.lookups, m);

	// Sort materials
	thrust::sort_by_key(thrust::device, GSD.mat_samples, GSD.mat_samples + in.lookups, GSD.p_energy_samples);
	
	// Launch all material kernels individually
	int offset = 0;
	for( int m = 0; m < 12; m++ )
	{
		nthreads = 32;
		nblocks = ceil((double) n_lookups_per_material[m] / (double) nthreads);
		xs_lookup_kernel_optimization_4<<<nblocks, nthreads>>>( in, GSD, m, n_lookups_per_material[m], offset );
		offset += n_lookups_per_material[m];
	}
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	
	////////////////////////////////////////////////////////////////////////////////
	// Reduce Verification Results
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Reducing verification results...\n");

	unsigned long verification_scalar = thrust::reduce(thrust::device, GSD.verification, GSD.verification + in.lookups, 0);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	return verification_scalar;
}

__global__ void xs_lookup_kernel_optimization_4(Inputs in, SimulationData GSD, int m, int n_lookups, int offset )
{
	// The lookup ID. Used to set the seed, and to store the verification value
	int i = blockIdx.x *blockDim.x + threadIdx.x;

	if( i >= n_lookups )
		return;

	i += offset;

	// Check that our material type matches the kernel material
	int mat = GSD.mat_samples[i];
	if( mat != m )
		return;

	double macro_xs_vector[5] = {0};
		
	// Perform macroscopic Cross Section Lookup
	calculate_macro_xs(
			GSD.p_energy_samples[i],        // Sampled neutron energy (in lethargy)
			mat,             // Sampled material type index neutron is in
			in.n_isotopes,   // Total number of isotopes in simulation
			in.n_gridpoints, // Number of gridpoints per isotope in simulation
			GSD.num_nucs,     // 1-D array with number of nuclides per material
			GSD.concs,        // Flattened 2-D array with concentration of each nuclide in each material
			GSD.unionized_energy_array, // 1-D Unionized energy array
			GSD.index_grid,   // Flattened 2-D grid holding indices into nuclide grid for each unionized energy level
			GSD.nuclide_grid, // Flattened 2-D grid holding energy levels and XS_data for all nuclides in simulation
			GSD.mats,         // Flattened 2-D array with nuclide indices defining composition of each type of material
			macro_xs_vector, // 1-D array with result of the macroscopic cross section (5 different reaction channels)
			in.grid_type,    // Lookup type (nuclide, hash, or unionized)
			in.hash_bins,    // Number of hash bins used (if using hash lookup type)
			GSD.max_num_nucs  // Maximum number of nuclides present in any material
			);

	// For verification, and to prevent the compiler from optimizing
	// all work out, we interrogate the returned macro_xs_vector array
	// to find its maximum value index, then increment the verification
	// value by that index. In this implementation, we have each thread
	// write to its thread_id index in an array, which we will reduce
	// with a thrust reduction kernel after the main simulation kernel.
	double max = -1.0;
	int max_idx = 0;
	for(int j = 0; j < 5; j++ )
	{
		if( macro_xs_vector[j] > max )
		{
			max = macro_xs_vector[j];
			max_idx = j;
		}
	}
	GSD.verification[i] = max_idx+1;
}

////////////////////////////////////////////////////////////////////////////////////
// Optimization 5 -- Kernel Splitting + Fuel/Other Lookups + Fuel/Other Partition
////////////////////////////////////////////////////////////////////////////////////
// This optimization is similar to optimization 4, but instead of sorting
// fully by material, we just sort by fuel or not fuel. Similarly, instead of
// launching kernels for all materials, similar to optimization 3 we only launch
// kernels for the fuel and other mateirals.
////////////////////////////////////////////////////////////////////////////////////

// Comparator for partitioning stage
struct is_mat_fuel{
	__host__ __device__
		bool operator()(const int & a)
		{
			return a == 0;
		}
};

unsigned long long run_event_based_simulation_optimization_5(Inputs in, SimulationData GSD, int mype)
{
	const char * optimization_name = "Optimization 5 - Fuel/No Fuel Lookup Kernels + Fuel/No Fuel Sort";
	
	if( mype == 0)	printf("Simulation Kernel:\"%s\"\n", optimization_name);
	
	////////////////////////////////////////////////////////////////////////////////
	// Allocate Additional Data Structures Needed by Optimized Kernel
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Allocating additional device data required by kernel...\n");
	size_t sz;
	size_t total_sz = 0;

	sz = in.lookups * sizeof(double);
	gpuErrchk( hipMalloc((void **) &GSD.p_energy_samples, sz) );
	total_sz += sz;
	GSD.length_p_energy_samples = in.lookups;

	sz = in.lookups * sizeof(int);
	gpuErrchk( hipMalloc((void **) &GSD.mat_samples, sz) );
	total_sz += sz;
	GSD.length_mat_samples = in.lookups;
	
	if( mype == 0)	printf("Allocated an additional %.0lf MB of data on GPU.\n", total_sz/1024.0/1024.0);

	////////////////////////////////////////////////////////////////////////////////
	// Configure & Launch Simulation Kernel
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Beginning optimized simulation...\n");

	int nthreads = 32;
	int nblocks = ceil( (double) in.lookups / 32.0);
	
	sampling_kernel<<<nblocks, nthreads>>>( in, GSD );
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	// Count the number of fuel material lookups that need to be performed (fuel id = 0)
	int n_fuel_lookups = thrust::count(thrust::device, GSD.mat_samples, GSD.mat_samples + in.lookups, 0);

	// Partition fuel into the first part of the array
	thrust::partition(thrust::device, GSD.mat_samples, GSD.mat_samples + in.lookups, GSD.p_energy_samples, is_mat_fuel());

	// Launch all material kernels individually (asynchronous is allowed)
	nblocks = ceil( (double) n_fuel_lookups / (double) nthreads);
	xs_lookup_kernel_optimization_5<<<nblocks, nthreads>>>( in, GSD, n_fuel_lookups, 0 );

	nblocks = ceil( (double) (in.lookups - n_fuel_lookups) / (double) nthreads);
	xs_lookup_kernel_optimization_5<<<nblocks, nthreads>>>( in, GSD, in.lookups-n_fuel_lookups, n_fuel_lookups );

	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	
	////////////////////////////////////////////////////////////////////////////////
	// Reduce Verification Results
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Reducing verification results...\n");

	unsigned long verification_scalar = thrust::reduce(thrust::device, GSD.verification, GSD.verification + in.lookups, 0);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	return verification_scalar;
}

__global__ void xs_lookup_kernel_optimization_5(Inputs in, SimulationData GSD, int n_lookups, int offset )
{
	// The lookup ID. Used to set the seed, and to store the verification value
	int i = blockIdx.x *blockDim.x + threadIdx.x;

	if( i >= n_lookups )
		return;
	
	i += offset;

	double macro_xs_vector[5] = {0};
		
	// Perform macroscopic Cross Section Lookup
	calculate_macro_xs(
			GSD.p_energy_samples[i],        // Sampled neutron energy (in lethargy)
			GSD.mat_samples[i],             // Sampled material type index neutron is in
			in.n_isotopes,   // Total number of isotopes in simulation
			in.n_gridpoints, // Number of gridpoints per isotope in simulation
			GSD.num_nucs,     // 1-D array with number of nuclides per material
			GSD.concs,        // Flattened 2-D array with concentration of each nuclide in each material
			GSD.unionized_energy_array, // 1-D Unionized energy array
			GSD.index_grid,   // Flattened 2-D grid holding indices into nuclide grid for each unionized energy level
			GSD.nuclide_grid, // Flattened 2-D grid holding energy levels and XS_data for all nuclides in simulation
			GSD.mats,         // Flattened 2-D array with nuclide indices defining composition of each type of material
			macro_xs_vector, // 1-D array with result of the macroscopic cross section (5 different reaction channels)
			in.grid_type,    // Lookup type (nuclide, hash, or unionized)
			in.hash_bins,    // Number of hash bins used (if using hash lookup type)
			GSD.max_num_nucs  // Maximum number of nuclides present in any material
			);

	// For verification, and to prevent the compiler from optimizing
	// all work out, we interrogate the returned macro_xs_vector array
	// to find its maximum value index, then increment the verification
	// value by that index. In this implementation, we have each thread
	// write to its thread_id index in an array, which we will reduce
	// with a thrust reduction kernel after the main simulation kernel.
	double max = -1.0;
	int max_idx = 0;
	for(int j = 0; j < 5; j++ )
	{
		if( macro_xs_vector[j] > max )
		{
			max = macro_xs_vector[j];
			max_idx = j;
		}
	}
	GSD.verification[i] = max_idx+1;
}

////////////////////////////////////////////////////////////////////////////////////
// Optimization 6 -- Kernel Splitting + All Material Lookups + Full Sort
//                   + Energy Sort
////////////////////////////////////////////////////////////////////////////////////
// This optimization builds on optimization 4, adding in a second sort by energy.
// It is extremely fast, as now most of the threads within a warp will be hitting
// the same indices in the lookup grids. This greatly reduces thread divergence and
// greatly improves cache efficiency and re-use.
//
// However, it is unlikely that this exact optimization would be possible in a real
// application like OpenMC. One major difference is that particle objects are quite
// large, often having 50+ variable fields, such that sorting them in memory becomes
// rather expensive. Instead, the best possible option would probably be to create
// intermediate indexing (per Hamilton et. al 2019), and run the kernels indirectly.
////////////////////////////////////////////////////////////////////////////////////
unsigned long long run_event_based_simulation_optimization_6(Inputs in, SimulationData GSD, int mype)
{
	const char * optimization_name = "Optimization 6 - Material & Energy Sorts + Material-specific Kernels";
	
	if( mype == 0)	printf("Simulation Kernel:\"%s\"\n", optimization_name);
	
	////////////////////////////////////////////////////////////////////////////////
	// Allocate Additional Data Structures Needed by Optimized Kernel
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Allocating additional device data required by kernel...\n");
	size_t sz;
	size_t total_sz = 0;

	sz = in.lookups * sizeof(double);
	gpuErrchk( hipMalloc((void **) &GSD.p_energy_samples, sz) );
	total_sz += sz;
	GSD.length_p_energy_samples = in.lookups;

	sz = in.lookups * sizeof(int);
	gpuErrchk( hipMalloc((void **) &GSD.mat_samples, sz) );
	total_sz += sz;
	GSD.length_mat_samples = in.lookups;
	
	if( mype == 0)	printf("Allocated an additional %.0lf MB of data on GPU.\n", total_sz/1024.0/1024.0);

	////////////////////////////////////////////////////////////////////////////////
	// Configure & Launch Simulation Kernel
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Beginning optimized simulation...\n");

	int nthreads = 32;
	int nblocks = ceil( (double) in.lookups / 32.0);
	
	sampling_kernel<<<nblocks, nthreads>>>( in, GSD );
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	// Count the number of fuel material lookups that need to be performed (fuel id = 0)
	int n_lookups_per_material[12];
	for( int m = 0; m < 12; m++ )
		n_lookups_per_material[m] = thrust::count(thrust::device, GSD.mat_samples, GSD.mat_samples + in.lookups, m);

	// Sort by material first
	thrust::sort_by_key(thrust::device, GSD.mat_samples, GSD.mat_samples + in.lookups, GSD.p_energy_samples);

	// Now, sort each material by energy
	int offset = 0;
	for( int m = 0; m < 12; m++ )
	{
		thrust::sort_by_key(thrust::device, GSD.p_energy_samples + offset, GSD.p_energy_samples + offset + n_lookups_per_material[m], GSD.mat_samples + offset);
		offset += n_lookups_per_material[m];
	}
	
	// Launch all material kernels individually
	offset = 0;
	for( int m = 0; m < 12; m++ )
	{
		nthreads = 32;
		nblocks = ceil((double) n_lookups_per_material[m] / (double) nthreads);
		xs_lookup_kernel_optimization_4<<<nblocks, nthreads>>>( in, GSD, m, n_lookups_per_material[m], offset );
		offset += n_lookups_per_material[m];
	}
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	
	////////////////////////////////////////////////////////////////////////////////
	// Reduce Verification Results
	////////////////////////////////////////////////////////////////////////////////
	if( mype == 0)	printf("Reducing verification results...\n");

	unsigned long verification_scalar = thrust::reduce(thrust::device, GSD.verification, GSD.verification + in.lookups, 0);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	return verification_scalar;
}
