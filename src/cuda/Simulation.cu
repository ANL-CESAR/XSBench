#include "hip/hip_runtime.h"
#include "XSbench_header.h"

unsigned long long run_event_based_simulation(Inputs in, SimulationData SD, int mype)
{
	if( mype == 0)	
		printf("Beginning event based simulation...\n");
	
	////////////////////////////////////////////////////////////////////////////////
	// SUMMARY: Simulation Data Structure Manifest for "SD" Object
	// Here we list all heap arrays (and lengths) in SD that would need to be
	// offloaded manually if using an accelerator with a seperate memory space
	////////////////////////////////////////////////////////////////////////////////
	// int * num_nucs;                     // Length = length_num_nucs;
	// double * concs;                     // Length = length_concs
	// int * mats;                         // Length = length_mats
	// double * unionized_energy_array;    // Length = length_unionized_energy_array
	// int * index_grid;                   // Length = length_index_grid
	// NuclideGridPoint * nuclide_grid;    // Length = length_nuclide_grid
	// 
	// Note: "unionized_energy_array" and "index_grid" can be of zero length
	//        depending on lookup method.
	//
	// Note: "Lengths" are given as the number of objects in the array, not the
	//       number of bytes.
	////////////////////////////////////////////////////////////////////////////////


	////////////////////////////////////////////////////////////////////////////////
	// Begin Actual Simulation Loop 
	////////////////////////////////////////////////////////////////////////////////
	unsigned long long verification = 0;
	#pragma omp parallel for schedule(guided) reduction(+:verification)
	for( int i = 0; i < in.lookups; i++ )
	{
		// Particles are seeded by their particle ID
		unsigned long seed = ((unsigned long) i+ (unsigned long)1)* (unsigned long) 13371337;

		// Randomly pick an energy and material for the particle
		double p_energy = rn(&seed);
		int mat      = pick_mat(&seed); 

		// debugging
		//printf("E = %lf mat = %d\n", p_energy, mat);

		double macro_xs_vector[5] = {0};

		// Perform macroscopic Cross Section Lookup
		calculate_macro_xs(
				p_energy,        // Sampled neutron energy (in lethargy)
				mat,             // Sampled material type index neutron is in
				in.n_isotopes,   // Total number of isotopes in simulation
				in.n_gridpoints, // Number of gridpoints per isotope in simulation
				SD.num_nucs,     // 1-D array with number of nuclides per material
				SD.concs,        // Flattened 2-D array with concentration of each nuclide in each material
				SD.unionized_energy_array, // 1-D Unionized energy array
				SD.index_grid,   // Flattened 2-D grid holding indices into nuclide grid for each unionized energy level
				SD.nuclide_grid, // Flattened 2-D grid holding energy levels and XS_data for all nuclides in simulation
				SD.mats,         // Flattened 2-D array with nuclide indices defining composition of each type of material
				macro_xs_vector, // 1-D array with result of the macroscopic cross section (5 different reaction channels)
				in.grid_type,    // Lookup type (nuclide, hash, or unionized)
				in.hash_bins,    // Number of hash bins used (if using hash lookup type)
				SD.max_num_nucs  // Maximum number of nuclides present in any material
				);

		// For verification, and to prevent the compiler from optimizing
		// all work out, we interrogate the returned macro_xs_vector array
		// to find its maximum value index, then increment the verification
		// value by that index. In this implementation, we prevent thread
		// contention by using an OMP reduction on the verification value.
		// For accelerators, a different approach might be required
		// (e.g., atomics, reduction of thread-specific values in large
		// array via CUDA thrust, etc).
		double max = -1.0;
		int max_idx = 0;
		for(int i = 0; i < 5; i++ )
		{
			if( macro_xs_vector[i] > max )
			{
				max = macro_xs_vector[i];
				max_idx = i;
			}
		}
		verification += max_idx;
	}

	return verification;
}

unsigned long long run_history_based_simulation(Inputs in, SimulationData SD, int mype)
{
	if( mype == 0)	
		printf("Beginning history based simulation...\n");

	
	////////////////////////////////////////////////////////////////////////////////
	// SUMMARY: Simulation Data Structure Manifest for "SD" Object
	// Here we list all heap arrays (and lengths) in SD that would need to be
	// offloaded manually if using an accelerator with a seperate memory space
	////////////////////////////////////////////////////////////////////////////////
	// int * num_nucs;                     // Length = length_num_nucs;
	// double * concs;                     // Length = length_concs
	// int * mats;                         // Length = length_mats
	// double * unionized_energy_array;    // Length = length_unionized_energy_array
	// int * index_grid;                   // Length = length_index_grid
	// NuclideGridPoint * nuclide_grid;    // Length = length_nuclide_grid
	// 
	// Note: "unionized_energy_array" and "index_grid" can be of zero length
	//        depending on lookup method.
	//
	// Note: "Lengths" are given as the number of objects in the array, not the
	//       number of bytes.
	////////////////////////////////////////////////////////////////////////////////

	unsigned long long verification = 0;

	// Begin outer lookup loop over particles. This loop is independent.
	#pragma omp parallel for schedule(guided) reduction(+:verification)
	for( int p = 0; p < in.particles; p++ )
	{
		// Particles are seeded by their particle ID
		unsigned long seed = ((unsigned long) p+ (unsigned long)1)* (unsigned long) 13371337;

		// Randomly pick an energy and material for the particle
		double p_energy = rn(&seed);
		int mat      = pick_mat(&seed); 

		// Inner XS Lookup Loop
		// This loop is dependent!
		// i.e., Next iteration uses data computed in previous iter.
		for( int i = 0; i < in.lookups; i++ )
		{
			// debugging
			//printf("E = %lf mat = %d\n", p_energy, mat);

			double macro_xs_vector[5] = {0};

			// Perform macroscopic Cross Section Lookup
			calculate_macro_xs(
					p_energy,        // Sampled neutron energy (in lethargy)
					mat,             // Sampled material type neutron is in
					in.n_isotopes,   // Total number of isotopes in simulation
					in.n_gridpoints, // Number of gridpoints per isotope in simulation
					SD.num_nucs,     // 1-D array with number of nuclides per material
					SD.concs,        // Flattened 2-D array with concentration of each nuclide in each material
					SD.unionized_energy_array, // 1-D Unionized energy array
					SD.index_grid,   // Flattened 2-D grid holding indices into nuclide grid for each unionized energy level
					SD.nuclide_grid, // Flattened 2-D grid holding energy levels and XS_data for all nuclides in simulation
					SD.mats,         // Flattened 2-D array with nuclide indices for each type of material
					macro_xs_vector, // 1-D array with result of the macroscopic cross section (5 different reaction channels)
					in.grid_type,    // Lookup type (nuclide, hash, or unionized)
					in.hash_bins,    // Number of hash bins used (if using hash lookups)
					SD.max_num_nucs  // Maximum number of nuclides present in any material
					);

		
			// For verification, and to prevent the compiler from optimizing
			// all work out, we interrogate the returned macro_xs_vector array
			// to find its maximum value index, then increment the verification
			// value by that index. In this implementation, we prevent thread
			// contention by using an OMP reduction on it. For other accelerators,
			// a different approach might be required (e.g., atomics, reduction
			// of thread-specific values in large array via CUDA thrust, etc)
			double max = -1.0;
			int max_idx = 0;
			for(int j = 0; j < 5; j++ )
			{
				if( macro_xs_vector[j] > max )
				{
					max = macro_xs_vector[j];
					max_idx = j;
				}
			}
			verification += max_idx;

			// Randomly pick next energy and material for the particle
			// Also incorporates results from macro_xs lookup to
			// enforce loop dependency.
			// In a real MC app, this dependency is expressed in terms
			// of branching physics sampling, whereas here we are just
			// artificially enforcing this dependence based on altering
			// the seed
			for( int j = 0; j < 5; j++ )
				seed += macro_xs_vector[j] * (j+1)*1337*1337;

			p_energy = rn(&seed);
			mat      = pick_mat(&seed); 
		}

	}
	return verification;
}
