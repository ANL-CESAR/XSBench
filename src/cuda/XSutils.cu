#include "hip/hip_runtime.h"
#include "XSbench_header.cuh"

int double_compare(const void * a, const void * b)
{
	double A = *((double *) a);
	double B = *((double *) b);

	if( A > B )
		return 1;
	else if( A < B )
		return -1;
	else
		return 0;
}

int NGP_compare(const void * a, const void * b)
{
	NuclideGridPoint A = *((NuclideGridPoint *) a);
	NuclideGridPoint B = *((NuclideGridPoint *) b);

	if( A.energy > B.energy )
		return 1;
	else if( A.energy < B.energy )
		return -1;
	else
		return 0;
}


// RNG Used for Verification Option.
// This one has a static seed (must be set manually in source).
// Park & Miller Multiplicative Conguential Algorithm
// From "Numerical Recipes" Second Edition
double rn_v(void)
{
	static unsigned long seed = 1337;
	double ret;
	unsigned long n1;
	unsigned long a = 16807;
	unsigned long m = 2147483647;
	n1 = ( a * (seed) ) % m;
	seed = n1;
	ret = (double) n1 / m;
	return ret;
}


size_t estimate_mem_usage( Inputs in )
{
	size_t single_nuclide_grid = in.n_gridpoints * sizeof( NuclideGridPoint );
	size_t all_nuclide_grids   = in.n_isotopes * single_nuclide_grid;
	size_t size_UEG            = in.n_isotopes*in.n_gridpoints*sizeof(double) + in.n_isotopes*in.n_gridpoints*in.n_isotopes*sizeof(int);
	size_t size_hash_grid      = in.hash_bins * in.n_isotopes * sizeof(int);
	size_t memtotal;

	if( in.grid_type == UNIONIZED )
		memtotal          = all_nuclide_grids + size_UEG;
	else if( in.grid_type == NUCLIDE )
		memtotal          = all_nuclide_grids;
	else
		memtotal          = all_nuclide_grids + size_hash_grid;

	memtotal          = ceil(memtotal / (1024.0*1024.0));
	return memtotal;
}

double get_time(void)
{
	#ifdef MPI
	return MPI_Wtime();
	#endif

	#ifdef OPENMP
	return omp_get_wtime();
	#endif

	// If using C++, we can do this:
	unsigned long us_since_epoch = std::chrono::high_resolution_clock::now().time_since_epoch() / std::chrono::microseconds(1);
	return (double) us_since_epoch / 1.0e6;
}
